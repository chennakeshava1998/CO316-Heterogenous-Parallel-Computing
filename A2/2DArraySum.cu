#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <math.h>

#define M 512
#define N 512

__global__ void add(int *A, int *B, int *C)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col < N && row < M)
        C[row * N + col] = A[row * N + col] + B[row * N + col];
}

__global__ void singleThreadVecAdd(int *A, int *B, int *C)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
            C[i * N + j] = A[i * N + j] + B[i * N + j];
    }
}

int main()
{
    printf("\n\nProgram to perform Vector Addition in CUDA\n\n");

    int *A, *B, *C;
    int host_A[M][N], host_B[M][N], host_C[M][N];

    // generate random int numbers for input
    printf("\nGenerating %d int numbers for the input arrays....\n", N * M);
    int i,j;
    for (i = 0; i < M; ++i)
    {
        for (j = 0; j < N; ++j)
            host_A[i][j] = sin(i) + sin(j);
//            host_A[i][j] = 1.0;


    }

    for (i = 0; i < M; ++i)
    {
        for (j = 0; j < N; ++j)
            host_B[i][j] = 1.0;
    }

    printf("\nAllocating memory on the GPU...\n\n");
    // allocate space on device
    hipMalloc((void **)&A, M * N * sizeof(int));
    hipMalloc((void **)&B, M * N * sizeof(int));
    hipMalloc((void **)&C, M * N * sizeof(int));

    // memory transfer from host to device
    printf("\nTransferring data from host to device for computations...\n\n");

    hipMemcpy(A, host_A, M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(B, host_B, M * N * sizeof(int), hipMemcpyHostToDevice);

    // dimensions of thread block + kernel launch
    dim3 blockDim(16, 16, 1);

    dim3 gridDim((int)ceil((float)(M) / blockDim.x),(float) ceil((int)(N) / blockDim.y), 1);

    printf("\n\nCalling the kernel with %d Blocks and %d threads in each block\n", gridDim, blockDim);

    // timing the GPU kernel
    double t1 = clock();

    add<<<gridDim, blockDim>>>(A, B, C);
    hipDeviceSynchronize();
    double t2 = clock();

    printf("\nNumber of threads per block: %d\n", blockDim.x * blockDim.y);
    printf("\nDimesions of the grid: %d BY %d BY %d\n", gridDim.x, gridDim.y, gridDim.z);
    printf("\nTime taken to add %d elements = %lf\n\n", M * N, (t2 - t1) / CLOCKS_PER_SEC);

    // copy back to host
    printf("\n\nCalculation completed on the GPU. Fetching the answer back from the GPU's global memory\n");
    hipMemcpy(host_C, C, M * N * sizeof(int), hipMemcpyDeviceToHost);

    // Calculating the time required for a single thread, within a single block
    t1 = clock();
    singleThreadVecAdd<<<1, 1>>>(A, B, C);
    hipDeviceSynchronize();
    t2 = clock();

    // copy back to host
    hipMemcpy(host_C, C, M * N * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nTime taken to perform %d additions with single thread and One block: %lf\n", M * N, (t2 - t1) / CLOCKS_PER_SEC);

    // free the malloc'ed memory
    printf("\n\nFree'ing the malloc'ed memory on the GPU\n");
    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}
