#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <math.h>

#define M 1024
#define N 4096

__global__ void add(float **A, float **B, float **C)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col < N && row < M)
        C[row][col] = A[row][col] + B[row][col];
}

__global__ void singleThreadVecAdd(float **A, float **B, float **C)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
            C[i][j] = A[i][j] + B[i][j];
    }
}

int main()
{
    printf("\n\nProgram to perform Vector Addition in CUDA\n\n");

    float **A, **B, **C;
    float host_A[M][N], host_B[M][N], host_C[M][N];

    // generate random floating numbers for input
    printf("\nGenerating %d floating-point numbers for the input arrays....\n", N * M);

    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
            host_A[i][j] = sin(i) + sin(j);
    }

    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
            host_B[i][j] = cos(i) + cos(j);
    }

    printf("\nAllocating memory on the GPU...\n\n");
    // allocate space on device
    hipMalloc((void **)&A, M * N * sizeof(float));
    hipMalloc((void **)&B, M * N * sizeof(float));
    hipMalloc((void **)&C, M * N * sizeof(float));

    // memory transfer from host to device
    printf("\nTransferring data from host to device for computations...\n\n");

    hipMemcpy(A, host_A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B, host_B, M * N * sizeof(float), hipMemcpyHostToDevice);

    // dimensions of thread block + kernel launch
    int blockDim = 1024;

    dim3 gridDim(ceil((float)(M) / 1024), ceil((float)(N) / 1024), 1);

    // printf("\n\nCalling the kernel with %d Blocks and %d threads in each block\n", gridDim, blockDim);

    // timing the GPU kernel
    double t1 = clock();

    add<<<gridDim, blockDim>>>(A, B, C);
    hipDeviceSynchronize();
    double t2 = clock();

    printf("\nNumber of threads per block: %d\n", blockDim);
    printf("\nDimesions of the grid: %d BY %d BY %d\n", gridDim.x, gridDim.y, gridDim.z);
    printf("\nTime taken to add %d elements = %lf\n\n", M * N, (t2 - t1) / CLOCKS_PER_SEC);

    // copy back to host
    printf("\n\nCalculation completed on the GPU. Fetching the answer back from the GPU's global memory\n");
    hipMemcpy(host_C, C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Calculating the time required for a single thread, within a single block
    t1 = clock();
    singleThreadVecAdd<<<1, 1>>>(A, B, C);
    hipDeviceSynchronize();
    t2 = clock();

    // copy back to host
    hipMemcpy(host_C, C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    printf("\nTime taken to perform %d additions with single thread and One block: %lf\n", M * N, (t2 - t1) / CLOCKS_PER_SEC);

    // free the malloc'ed memory
    printf("\n\nFree'ing the malloc'ed memory on the GPU\n");
    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}
