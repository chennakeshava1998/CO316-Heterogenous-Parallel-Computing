#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <math.h>

// Number of rows in the first input matrix
#define M 512

// Number of columns in the first input matrix
#define N 512

// Dimensions of the input matrices: (M, N) and (N, M)

__global__ void matMul(int *A, int *B, int *C)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col < M && row < M)
    {
        long long prod_val = 0;
        for (int k = 0; k < N; ++k)
            prod_val += (A[row * N + k] * B[k * N + col]);

        C[row * N + col] = prod_val;
    }
}

__global__ void singleThreadVecMul(int *A, int *B, int *C)
{
    for (int row = 0; row < M; ++row)
    {
        for (int col = 0; col < M; ++col)
        {
            long long prod_val = 0;
            for (int k = 0; k < N; ++k)
                prod_val += (A[row * N + k] * B[k * N + col]);

            C[row * N + col] = prod_val;
        }
    }
}

void CPUMatMul(int A[M][N], int B[N][M], int C[M][M])
{

    for (int row = 0; row < M; ++row)
    {
        for (int col = 0; col < M; ++col)
        {
            int prod_val = 0;
            for (int k = 0; k < N; ++k)
	    {
            	prod_val = prod_val + (A[row][k] * B[k][col]);
	    }
            C[row][col] = prod_val;
        }
    }

    
}

bool compare(int A[M][M], int B[M][M], double accuracy)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < M; ++j)
            if ((abs(A[i][j] - B[i][j])) > accuracy)
                return 0;
    }

    return 1;
}

int main()
{
    printf("\n\nProgram to perform Matrix Multiplication in CUDA\n\n");

    int *A, *B, *C;
    int host_A[M][N], host_B[N][M], host_C[M][M], CPUMatMulAns[M][M];

    // generate random int numbers for input
    printf("\nGenerating %d int numbers for the input arrays....\n", N * M);
    int i, j;
    for (i = 0; i < M; ++i)
    {
        for (j = 0; j < N; ++j)
            host_A[i][j] = sin(i) + sin(j);
    }

    for (i = 0; i < N; ++i)
    {
        for (j = 0; j < M; ++j)
            host_B[i][j] = cos(i) + cos(j);
    }

    CPUMatMul(host_A, host_B, CPUMatMulAns);

    printf("\nAllocating memory on the GPU...\n\n");
    // allocate space on device
    hipMalloc((void **)&A, M * N * sizeof(int));
    hipMalloc((void **)&B, M * N * sizeof(int));
    hipMalloc((void **)&C, M * M * sizeof(int));

    // memory transfer from host to device
    printf("\nTransferring data from host to device for computations...\n\n");

    hipMemcpy(A, host_A, M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(B, host_B, M * N * sizeof(int), hipMemcpyHostToDevice);

    // dimensions of thread block + kernel launch
    dim3 blockDim(16, 16, 1);

    dim3 gridDim((int)ceil((float)(M) / blockDim.x), (float)ceil((int)(N) / blockDim.y), 1);

    printf("\n\nCalling the kernel with %d Blocks and %d threads in each block\n", gridDim.x * gridDim.y, blockDim.x * blockDim.y);

    // timing the GPU kernel
    double t1 = clock();

    matMul<<<gridDim, blockDim>>>(A, B, C);
    hipDeviceSynchronize();
    double t2 = clock();

    // copy back to host
    printf("\n\nCalculation completed on the GPU. Fetching the answer back from the GPU's global memory\n");
    hipMemcpy(host_C, C, M * M * sizeof(int), hipMemcpyDeviceToHost);

    // checking of the required accuracy is attained
    double accuracy = pow(10, -6);
    if (compare(CPUMatMulAns, host_C, accuracy))
        printf("The answers generated by GPU are within %lf accuracy\n\n", accuracy);

    else
        printf("The answers generated by GPU are NOT within %lf accuracy\n\n", accuracy);

    printf("\nNumber of threads per block: %d\n", blockDim.x * blockDim.y);
    printf("\nDimesions of the grid: %d BY %d BY %d\n", gridDim.x, gridDim.y, gridDim.z);
    printf("\nTotal Time taken for %ld operations = %lf\n\n", M * M * N, (t2 - t1) / CLOCKS_PER_SEC);

    // Calculating the time required for a single thread, within a single block
    t1 = clock();
    singleThreadVecMul<<<1, 1>>>(A, B, C);
    hipDeviceSynchronize();
    t2 = clock();

    // copy back to host
    hipMemcpy(host_C, C, M * M * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nTime taken to perform %ld additions with single thread and One block: %lf\n", M * M * N, (t2 - t1) / CLOCKS_PER_SEC);

    // free the malloc'ed memory
    printf("\n\nFree'ing the malloc'ed memory on the GPU\n");
    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}
